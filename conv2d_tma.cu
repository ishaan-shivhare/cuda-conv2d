#include "hip/hip_runtime.h"
#include "conv2d_config.h" 
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cudaTypedefs.h>  // PFN_cuTensorMapEncodeTiled
#include <torch/extension.h>

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

#define CUDA_CHECK(err) do { \
    hipError_t err_ = err; \
    if (err_ != hipSuccess) { \
        printf("CUDA error %s at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
        exit(1); \
    } \
} while (0)

#define CU_CHECK(err) do { \
    hipError_t err_ = err; \
    if (err_ != hipSuccess) { \
        const char* errStr; \
        hipDrvGetErrorString(err_, &errStr); \
        printf("Driver API error: %s at %s:%d\n", errStr, __FILE__, __LINE__); \
        exit(1); \
    } \
} while (0)


PFN_cuTensorMapEncodeTiled_v12000 get_cuTensorMapEncodeTiled() {
    void* func_ptr = nullptr;
    hipDriverEntryPointQueryResult status;
    CUDA_CHECK(cudaGetDriverEntryPointByVersion(
        "cuTensorMapEncodeTiled", &func_ptr, 12000, hipEnableDefault, &status));
    assert(status == hipDriverEntryPointSuccess);
    return reinterpret_cast<PFN_cuTensorMapEncodeTiled_v12000>(func_ptr);
}

// Wraps input and output CUtensorMap creation
void setup_tensor_maps(CUtensorMap& input_map, CUtensorMap& output_map,
                       void* d_input, void* d_output,
                       int IN_C, int H, int W,
                       int padded_out_h, int padded_out_w,
                       int BLOCK_SIZE) {
    constexpr uint32_t RANK = 3;
    constexpr uint32_t RANK_OUT = 2;

    uint64_t input_dims[RANK] = {static_cast<uint64_t>(W), static_cast<uint64_t>(H), static_cast<uint64_t>(IN_C)};
    uint64_t input_strides[RANK - 1] = {
        static_cast<uint64_t>(W * sizeof(float)),
        static_cast<uint64_t>(H * W * sizeof(float))
    };
    uint32_t input_box[RANK] = {static_cast<uint32_t>(TILE_W_PAD), static_cast<uint32_t>(SH_TILE_W), 1};
    uint32_t input_elem_stride[RANK] = {1, 1, 1};

    uint64_t output_dims[RANK_OUT] = {static_cast<uint64_t>(padded_out_w), static_cast<uint64_t>(padded_out_h)};
    uint64_t output_strides[RANK_OUT - 1] = {
        static_cast<uint64_t>(padded_out_w* sizeof(float))
    };
    uint32_t output_box[RANK_OUT] = {static_cast<uint32_t>(BLOCK_SIZE), static_cast<uint32_t>(BLOCK_SIZE)};
    uint32_t output_elem_stride[RANK_OUT] = {1, 1};

    auto encode = get_cuTensorMapEncodeTiled();

    hipError_t res = encode(
        &input_map,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        RANK,
        d_input,
        input_dims,
        input_strides,
        input_box,
        input_elem_stride,
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    // assert(res == hipSuccess);
    if (res != hipSuccess) {
        printf("cuTensorMapEncodeTiled for input_map failed with error code: %d\n", res);
        std::abort();
    }


    res = encode(
        &output_map,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        RANK_OUT,
        d_output,
        output_dims,
        output_strides,
        output_box,
        output_elem_stride,
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    if (res != hipSuccess) {
        printf("cuTensorMapEncodeTiled for output_map failed with error code: %d\n", res);
        std::abort();
    }
}

__global__ void producer_consumer_pattern(
    const __grid_constant__ CUtensorMap input_map,
    const __grid_constant__ CUtensorMap output_map,
    float* kernel,
    float* out,
    int OUT_H,
    int OUT_W,
    float* inp,
    int H,
    int W,
    int padded_out_w
) {

    // Shared memory buffer declared below is of size 2 * buffer_len
    // so that we can alternatively work between two buffers.
    // TODO: Maybe merge into one big buffer of size some constant instead of separate 
    // Shared tiles
    __shared__ alignas(128) float smem_buf0[SH_TILE_W][TILE_W_PAD];
    __shared__ alignas(128) float smem_buf1[SH_TILE_W][TILE_W_PAD];
    __shared__ alignas(128) float smem_out[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float smem_kernel[IN_C][K][K];
    __shared__ barrier bar_ready[2]; // track if buffers buffer_0 and buffer_1 are ready to be filled,
    __shared__ barrier bar_filled[2]; // track if buffers buffer_0 and buffer_1 are filled-in respectively

    int tid = threadIdx.y*blockDim.x + threadIdx.x;
    if (tid < 2) {
        init(bar_ready + tid, blockDim.x * blockDim.y);
        init(bar_filled + tid, blockDim.x * blockDim.y);
        cde::fence_proxy_async_shared_cta();
    }
    
    // Load kernel into SMEM
    for (int i = tid; i < IN_C * K * K; i += blockDim.x * blockDim.y) {
        int c  = i / (K*K);
        int kk = i % (K*K);
        int ky = kk / K, kx = kk % K;
        smem_kernel[c][ky][kx] = kernel[c*K*K + ky*K + kx];
    }
    
    // Zero out smem_out
    for (int i = tid; i < BLOCK_SIZE*BLOCK_SIZE; i += blockDim.x * blockDim.y) {
        int y = i / BLOCK_SIZE, x = i % BLOCK_SIZE;
        smem_out[y][x] = 0.0f;
    }
    
    // Sync to make barriers, kernel, zeroed out accumulator visible to everyone
    __syncthreads();

    // For now: 1 producer warpgroup and 1 consumer warpgroup (16x16 block)
    // Can increase block size to 16x32 for 3 consumer 1 producer
    if (tid < warpSize * 4) {
        // consumer
        barrier::arrival_token token1 = bar_ready[0].arrive(); // buffer 0 is ready for initial fill 
        barrier::arrival_token token2 = bar_ready[1].arrive(); // buffer 1 is ready for initial fill 
        for (int c = 0; c < IN_C; ++c) {
            bar_filled[c%2].arrive_and_wait();
            float (*cur_buf)[TILE_W_PAD] = (c%2==0) ? smem_buf0 : smem_buf1;
            for (int i = tid; i < BLOCK_SIZE*BLOCK_SIZE; i += warpSize * 4) {
                int local_y = i / BLOCK_SIZE;
                int local_x = i % BLOCK_SIZE;
                float accum = 0.0f;
                for (int ky = 0; ky < K; ++ky) {
                    for (int kx = 0; kx < K; ++kx) {
                        int sy = local_y + ky;
                        int sx = local_x + kx;
                        if (sy < SH_TILE_W && sx < SH_TILE_W) {
                            accum += cur_buf[sy][sx] * smem_kernel[c][ky][kx];
                        }
                    }
                }
                smem_out[local_y][local_x] += accum;
            }
            barrier::arrival_token token = bar_ready[c%2].arrive();
        }

    }
    else {
        // producer
        // TODO: give up registers with "setmaxnreg" inline PTX 
        for (int c = 0; c < IN_C; ++c) {
            // Determine which buffer to load into
            float (*cur_buf)[TILE_W_PAD] = (c%2==0) ? smem_buf0 : smem_buf1;
            bar_ready[c%2].arrive_and_wait();
            barrier::arrival_token t_load;
            if (tid == 128) {
                // printf("reading channel %d from global for block (%d, %d) \n", c, blockIdx.x, blockIdx.y);
                cde::cp_async_bulk_tensor_3d_global_to_shared(
                    &cur_buf[0][0],
                    &input_map,
                    blockIdx.x*BLOCK_SIZE,
                    blockIdx.y*BLOCK_SIZE,
                    c,
                    bar_filled[c%2]
                );
                t_load = cuda::device::barrier_arrive_tx(bar_filled[c%2], 1, SH_TILE_W * TILE_W_PAD * sizeof(float));
            }
            else {
                t_load = bar_filled[c%2].arrive();
            }
        }
    }
    
    cde::fence_proxy_async_shared_cta();
    __syncthreads();
    // Final TMA store of smem_out → global
    if (tid == 0) {
        // printf("Committing output tile for block (%d, %d)\n", blockIdx.x, blockIdx.y);
        cde::cp_async_bulk_tensor_2d_shared_to_global(
            &output_map,
            blockIdx.x*BLOCK_SIZE,
            blockIdx.y*BLOCK_SIZE,
            &smem_out[0][0]
        );
        cde::cp_async_bulk_commit_group();
        cde::cp_async_bulk_wait_group_read<0>();
    }
}

void launch_conv2d_tma(torch::Tensor input, torch::Tensor kernel, torch::Tensor output, int padded_out_h, int padded_out_w) {
    TORCH_CHECK(input.device().is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(kernel.device().is_cuda(), "kernel must be a CUDA tensor");
    TORCH_CHECK(output.device().is_cuda(), "output must be a CUDA tensor");

    TORCH_CHECK(input.dtype() == torch::kFloat32, "input must be float32");
    TORCH_CHECK(kernel.dtype() == torch::kFloat32, "kernel must be float32");
    TORCH_CHECK(output.dtype() == torch::kFloat32, "output must be float32");

    TORCH_CHECK(input.dim() == 3, "input must be [C, H, W]");
    TORCH_CHECK(kernel.dim() == 3, "kernel must be [C, K, K]");
    TORCH_CHECK(output.dim() == 2, "output must be [H_out, W_out]");

    TORCH_CHECK(input.size(0) == IN_C, "IN_C mismatch");
    TORCH_CHECK(kernel.size(0) == IN_C, "IN_C mismatch");
    TORCH_CHECK(kernel.size(1) == K && kernel.size(2) == K, "Kernel shape mismatch");

    int H = input.size(1);
    int W = input.size(2);
    int OUT_H = H - K + 1;
    int OUT_W = W - K + 1;

    // Setup tensor maps
    CUtensorMap input_map;
    CUtensorMap output_map;
    setup_tensor_maps(
        input_map, output_map,
        input.data_ptr<float>(), output.data_ptr<float>(),
        IN_C, H, W,
        padded_out_h, padded_out_w,
        BLOCK_SIZE
    );

    // Kernel launch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);  // z-dim needed due to TMA rules
    dim3 blocks(
        // (OUT_W + BLOCK_SIZE - 1) / BLOCK_SIZE,
        // (OUT_H + BLOCK_SIZE - 1) / BLOCK_SIZE
        H / BLOCK_SIZE, W / BLOCK_SIZE
    );

    producer_consumer_pattern<<<blocks, threads>>>(
        input_map,
        output_map,
        kernel.data_ptr<float>(),
        output.data_ptr<float>(),
        OUT_H,
        OUT_W,
        input.data_ptr<float>(),
        H,
        W,
        padded_out_w
    );
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel or TMA error: " << hipGetErrorString(err) << std::endl;
    }
}
