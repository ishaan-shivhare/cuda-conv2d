#include "hip/hip_runtime.h"
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cudaTypedefs.h>  // PFN_cuTensorMapEncodeTiled

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

PFN_cuTensorMapEncodeTiled_v12000 get_cuTensorMapEncodeTiled() {
    void* func_ptr = nullptr;
    hipDriverEntryPointQueryResult status;
    CUDA_CHECK(cudaGetDriverEntryPointByVersion(
        "cuTensorMapEncodeTiled", &func_ptr, 12000, hipEnableDefault, &status));
    assert(status == hipDriverEntryPointSuccess);
    return reinterpret_cast<PFN_cuTensorMapEncodeTiled_v12000>(func_ptr);
}

// Wraps input and output CUtensorMap creation
void setup_tensor_maps(CUtensorMap& input_map, CUtensorMap& output_map,
                       void* d_input, void* d_output,
                       int IN_C, int H, int W,
                       int OUT_H, int OUT_W,
                       int BLOCK_SIZE) {
    constexpr uint32_t RANK = 3;

    uint64_t input_dims[RANK] = {static_cast<uint64_t>(IN_C), static_cast<uint64_t>(H), static_cast<uint64_t>(W)};
    uint64_t input_strides[RANK - 1] = {
        static_cast<uint64_t>(H * W * sizeof(float)),
        static_cast<uint64_t>(W * sizeof(float))
    };
    uint32_t input_box[RANK] = {1, static_cast<uint32_t>(SH_TILE_W), static_cast<uint32_t>(SH_TILE_W)};
    uint32_t input_elem_stride[RANK] = {1, 1, 1};

    uint64_t output_dims[RANK] = {1, static_cast<uint64_t>(OUT_H), static_cast<uint64_t>(OUT_W)};
    uint64_t output_strides[RANK - 1] = {
        static_cast<uint64_t>(OUT_H * OUT_W * sizeof(float)),
        static_cast<uint64_t>(OUT_W * sizeof(float))
    };
    uint32_t output_box[RANK] = {1, static_cast<uint32_t>(BLOCK_SIZE), static_cast<uint32_t>(BLOCK_SIZE)};
    uint32_t output_elem_stride[RANK] = {1, 1, 1};

    auto encode = get_cuTensorMapEncodeTiled();

    hipError_t res = encode(
        &input_map,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT,
        RANK,
        d_input,
        input_dims,
        input_strides,
        input_box,
        input_elem_stride,
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    assert(res == hipSuccess);

    res = encode(
        &output_map,
        CU_TENSOR_MAP_DATA_TYPE_FLOAT,
        RANK,
        d_output,
        output_dims,
        output_strides,
        output_box,
        output_elem_stride,
        CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE,
        CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    assert(res == hipSuccess);
}


__device__ void producer(const __grid_constant__ CUtensorMap tensor_map, barrier ready[], barrier filled[], float* smem_buffer, int H, int W, int buffer_len)
{
    for (int c = 0; c < IN_C; ++c) {
        ready[c%2].arrive_and_wait(); /* wait for buffer_(c%2) to be ready to be filled */
        /* produce, i.e., fill in, buffer_(c%2)  */
        cde::cp_async_bulk_tensor_3d_global_to_shared(&smem_buffer[c % 2], &tensor_map, c, blockIdx.y * SH_TILE_W, blockIdx.x * SH_TILE_W, filled[c%2]);
        barrier::arrival_token token = cuda::device::barrier_arrive_tx(filled[c%2], 1, buffer_len * sizeof(float)); /* buffer_(c%2) is filled */
    }
}

__device__ void consumer(barrier ready[], barrier filled[], float* smem_buffer, float* smem_out, float* smem_kernel, int buffer_len)
{
    barrier::arrival_token token1 = ready[0].arrive(); /* buffer_0 is ready for initial fill */
    barrier::arrival_token token2 = ready[1].arrive(); /* buffer_1 is ready for initial fill */
    
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    if (tid == 0) return; // it is the producer thread
    
    int num_threads = blockDim.x * blockDim.y;
    int out_pixels = BLOCK_SIZE * BLOCK_SIZE;
    for (int i = tid - 1; i < out_pixels; i += num_threads - 1) {
        int out_y = i / BLOCK_SIZE;
        int out_x = i % BLOCK_SIZE;

        // Init local accumulator
        for (int c = 0; c < IN_C; ++c) {
            // float acc = 0.0f;
            int buf_idx = c % 2;
            barrier::arrival_token token = filled[buf_idx].arrive()
            filled[buf_idx].wait(std::move(token));
            /* consume buffer_(c%2) */
            for (int ky = 0; ky < K; ++ky) {
                for (int kx = 0; kx < K; ++kx) {
                    float val = smem_buffer[buf_idx][0][out_y + ky][out_x + kx];
                    float weight = smem_kernel[c][ky][kx];
                    smem_out[0][out_y][out_x] += val * weight;
                }
            }
            token = ready[buf_idx].arrive(); /* buffer_(c%2) is ready to be re-filled */
            
        }
    }   
}

//N is the total number of float elements in arrays in and out
__global__ void producer_consumer_pattern(int N, int buffer_len, float* in, float* out) {

    // Shared memory buffer declared below is of size 2 * buffer_len
    // so that we can alternatively work between two buffers.
    // buffer_0 = buffer and buffer_1 = buffer + buffer_len
    // __shared__ extern float buffer[];
    __shared__ alignas(128) float smem_buffer[2][1][SH_TILE_W][SH_TILE_W];

    __shared__ alignas(128) float smem_out[1][BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float smem_kernel[IN_C][K][K];

    // bar[0] and bar[1] track if buffers buffer_0 and buffer_1 are ready to be filled,
    // while bar[2] and bar[3] track if buffers buffer_0 and buffer_1 are filled-in respectively
    __shared__ barrier bar[4];
    
    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() < 2){
            init(bar + block.thread_rank(), block.size());
    } else if (block.thread_rank() < 4){
        init(bar + block.thread_rank(), 1);
    }  
    // block.sync();

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int total_weights = IN_C * K * K;
    for (int i = tid; i < total_weights; i += blockDim.x * blockDim.y) {
        int c = i / (K * K);
        int k = i % (K * K);
        int ky = k / K;
        int kx = k % K;
        smem_kernel[c][ky][kx] = kernel[c * K * K + ky * K + kx];
    }

    __syncthreads();
    
    if (tid == 0)
        producer(bar, bar+2, buffer, in, N, buffer_len);
    else
        consumer(bar, bar+2, buffer, out, N, buffer_len);
}