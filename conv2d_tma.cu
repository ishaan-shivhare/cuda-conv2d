#include "hip/hip_runtime.h"
#include "conv2d_config.h" 
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cudaTypedefs.h>  // PFN_cuTensorMapEncodeTiled
#include <torch/extension.h>

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

#define CUDA_CHECK(err) do { \
    hipError_t err_ = err; \
    if (err_ != hipSuccess) { \
        printf("CUDA error %s at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
        exit(1); \
    } \
} while (0)

#define CU_CHECK(err) do { \
    hipError_t err_ = err; \
    if (err_ != hipSuccess) { \
        const char* errStr; \
        hipDrvGetErrorString(err_, &errStr); \
        printf("Driver API error: %s at %s:%d\n", errStr, __FILE__, __LINE__); \
        exit(1); \
    } \
} while (0)


PFN_cuTensorMapEncodeTiled_v12000 get_cuTensorMapEncodeTiled() {
    void* func_ptr = nullptr;
    hipDriverEntryPointQueryResult status;
    CUDA_CHECK(cudaGetDriverEntryPointByVersion(
        "cuTensorMapEncodeTiled", &func_ptr, 12000, hipEnableDefault, &status));
    assert(status == hipDriverEntryPointSuccess);
    return reinterpret_cast<PFN_cuTensorMapEncodeTiled_v12000>(func_ptr);
}

// Wraps input and output CUtensorMap creation
void setup_tensor_maps(CUtensorMap& input_map, CUtensorMap& output_map,
                       void* d_input, void* d_output,
                       int IN_C, int H, int W,
                       int OUT_H, int OUT_W,
                       int BLOCK_SIZE) {
    constexpr uint32_t RANK = 3;

    uint64_t input_dims[RANK] = {static_cast<uint64_t>(IN_C), static_cast<uint64_t>(H), static_cast<uint64_t>(W)};
    uint64_t input_strides[RANK - 1] = {
        static_cast<uint64_t>(H * W * sizeof(float)),
        static_cast<uint64_t>(W * sizeof(float))
    };
    uint32_t input_box[RANK] = {1, static_cast<uint32_t>(SH_TILE_W), static_cast<uint32_t>(SH_TILE_W)};
    uint32_t input_elem_stride[RANK] = {1, 1, 1};

    uint64_t output_dims[RANK] = {1, static_cast<uint64_t>(OUT_H), static_cast<uint64_t>(OUT_W)};
    uint64_t output_strides[RANK - 1] = {
        static_cast<uint64_t>(OUT_H * OUT_W * sizeof(float)),
        static_cast<uint64_t>(OUT_W * sizeof(float))
    };
    uint32_t output_box[RANK] = {1, static_cast<uint32_t>(BLOCK_SIZE), static_cast<uint32_t>(BLOCK_SIZE)};
    uint32_t output_elem_stride[RANK] = {1, 1, 1};

    auto encode = get_cuTensorMapEncodeTiled();

    hipError_t res = encode(
        &input_map,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        RANK,
        d_input,
        input_dims,
        input_strides,
        input_box,
        input_elem_stride,
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    assert(res == hipSuccess);

    res = encode(
        &output_map,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT32,
        RANK,
        d_output,
        output_dims,
        output_strides,
        output_box,
        output_elem_stride,
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );
    assert(res == hipSuccess);
}


__device__ void producer(const CUtensorMap& tensor_map, barrier ready[], barrier filled[], float smem_buffer[2][1][SH_TILE_W][SH_TILE_W])
{
    for (int c = 0; c < IN_C; ++c) {
        ready[c%2].arrive_and_wait(); /* wait for buffer_(c%2) to be ready to be filled */
        /* produce, i.e., fill in, buffer_(c%2)  */
        cde::cp_async_bulk_tensor_3d_global_to_shared(&smem_buffer[c % 2], &tensor_map, c, blockIdx.y * SH_TILE_W, blockIdx.x * SH_TILE_W, filled[c%2]);
        barrier::arrival_token token = cuda::device::barrier_arrive_tx(filled[c%2], 1, SH_TILE_W * SH_TILE_W * sizeof(float)); /* buffer_(c%2) is filled */
    }
}

__device__ void consumer(barrier ready[], barrier filled[], float smem_buffer[2][1][SH_TILE_W][SH_TILE_W], float smem_out[1][BLOCK_SIZE][BLOCK_SIZE], float smem_kernel[IN_C][K][K])
{
    barrier::arrival_token token1 = ready[0].arrive(); /* buffer_0 is ready for initial fill */
    barrier::arrival_token token2 = ready[1].arrive(); /* buffer_1 is ready for initial fill */
    
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    if (tid == 0) return; // it is the producer thread
    
    int num_threads = blockDim.x * blockDim.y;
    int out_pixels = BLOCK_SIZE * BLOCK_SIZE;
    for (int i = tid - 1; i < out_pixels; i += num_threads - 1) {
        int out_y = i / BLOCK_SIZE;
        int out_x = i % BLOCK_SIZE;

        // Init local accumulator
        for (int c = 0; c < IN_C; ++c) {
            // float acc = 0.0f;
            int buf_idx = c % 2;
            barrier::arrival_token token = filled[buf_idx].arrive();
            filled[buf_idx].wait(std::move(token));
            /* consume buffer_(c%2) */
            for (int ky = 0; ky < K; ++ky) {
                for (int kx = 0; kx < K; ++kx) {
                    float val = smem_buffer[buf_idx][0][out_y + ky][out_x + kx];
                    float weight = smem_kernel[c][ky][kx];
                    smem_out[0][out_y][out_x] += val * weight;
                }
            }
            token = ready[buf_idx].arrive(); /* buffer_(c%2) is ready to be re-filled */
            
        }
    }   
}

//N is the total number of float elements in arrays in and out
__global__ void producer_consumer_pattern(const __grid_constant__ CUtensorMap input_map, const __grid_constant__ CUtensorMap output_map, float* kernel, float* out) {

    // Shared memory buffer declared below is of size 2 * buffer_len
    // so that we can alternatively work between two buffers.
    // buffer_0 = buffer and buffer_1 = buffer + buffer_len
    // __shared__ extern float buffer[];
    __shared__ alignas(128) float smem_buffer[2][1][SH_TILE_W][SH_TILE_W];

    __shared__ alignas(128) float smem_out[1][BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float smem_kernel[IN_C][K][K];

    // bar[0] and bar[1] track if buffers buffer_0 and buffer_1 are ready to be filled,
    // while bar[2] and bar[3] track if buffers buffer_0 and buffer_1 are filled-in respectively
    __shared__ barrier bar[4];
    
    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() < 2){
            init(bar + block.thread_rank(), block.size());  // ready
    } else if (block.thread_rank() < 4){
        init(bar + block.thread_rank(), 1); // filled
    }  
    // block.sync();

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int total_weights = IN_C * K * K;
    for (int i = tid; i < total_weights; i += blockDim.x * blockDim.y) {
        int c = i / (K * K);
        int k = i % (K * K);
        int ky = k / K;
        int kx = k % K;
        smem_kernel[c][ky][kx] = kernel[c * K * K + ky * K + kx];
    }

    // Zero out the output block which will hold partial accumulations
    if (tid < BLOCK_SIZE * BLOCK_SIZE) {
        int y = tid / BLOCK_SIZE;
        int x = tid % BLOCK_SIZE;
        smem_out[0][y][x] = 0.0f;
    }
    __syncthreads();
    
    if (tid == 0)
        producer(input_map, bar, bar+2, smem_buffer);
    else
        consumer(bar, bar+2, smem_buffer, smem_out, smem_kernel);

    __syncthreads();
    if (tid == 0) {
        cde::cp_async_bulk_tensor_3d_shared_to_global(&output_map, 0, blockIdx.y, blockIdx.x, &smem_out);
        cde::cp_async_bulk_commit_group();
        cde::cp_async_bulk_wait_group_read<0>();
    }

}

void launch_conv2d_tma(torch::Tensor input, torch::Tensor kernel, torch::Tensor output) {
    TORCH_CHECK(input.device().is_cuda(), "input must be a CUDA tensor");
    TORCH_CHECK(kernel.device().is_cuda(), "kernel must be a CUDA tensor");
    TORCH_CHECK(output.device().is_cuda(), "output must be a CUDA tensor");

    TORCH_CHECK(input.dtype() == torch::kFloat32, "input must be float32");
    TORCH_CHECK(kernel.dtype() == torch::kFloat32, "kernel must be float32");
    TORCH_CHECK(output.dtype() == torch::kFloat32, "output must be float32");

    TORCH_CHECK(input.dim() == 3, "input must be [C, H, W]");
    TORCH_CHECK(kernel.dim() == 3, "kernel must be [C, K, K]");
    // TORCH_CHECK(output.dim() == 2, "output must be [H_out, W_out]");

    TORCH_CHECK(input.size(0) == IN_C, "IN_C mismatch");
    TORCH_CHECK(kernel.size(0) == IN_C, "IN_C mismatch");
    TORCH_CHECK(kernel.size(1) == K && kernel.size(2) == K, "Kernel shape mismatch");

    int H = input.size(1);
    int W = input.size(2);
    int OUT_H = H - K + 1;
    int OUT_W = W - K + 1;

    // TORCH_CHECK(output.size(0) == OUT_H && output.size(1) == OUT_W, "Output shape mismatch");
    TORCH_CHECK(output.dim() == 3 &&
            output.size(0) == 1 &&
            output.size(1) == OUT_H &&
            output.size(2) == OUT_W,
            "Output shape must be [1, H_out, W_out]");

    // Setup tensor maps
    CUtensorMap input_map;
    CUtensorMap output_map;
    setup_tensor_maps(
        input_map, output_map,
        input.data_ptr<float>(), output.data_ptr<float>(),
        IN_C, H, W,
        OUT_H, OUT_W,
        BLOCK_SIZE
    );

    // Kernel launch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);  // z-dim not needed for this kernel
    dim3 blocks(
        (OUT_W + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (OUT_H + BLOCK_SIZE - 1) / BLOCK_SIZE
    );

    producer_consumer_pattern<<<blocks, threads>>>(
        input_map,
        output_map,
        kernel.data_ptr<float>(),
        output.data_ptr<float>()
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
    }
}
